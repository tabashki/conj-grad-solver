#include "hip/hip_runtime.h"
#include "parallel.h"
#include "utils.h"

#include <iostream>
#include <cassert>

using std::vector;

__constant__ int d_vec_size;

constexpr int BLOCK_SIZE = 256;


// CUDA kernel helper since there's no native float AtomicMax
__device__ void atomicAbsMaxFloat(float* dest, float value)
{
    // This *should* be safe when the sign bit is always zero
    atomicMax(reinterpret_cast<int*>(dest), __float_as_int(abs(value)));
}


// CUDA kernel that calculates a matrix-by-vector product
// where A is the sparse matrix A of 3 diagonals (1, -2, 1)
__global__ void parallel_sp_matmul(float* dest, const float* x_vec)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Hoping that these will be converted to conditional/masked loads
    const float left = (i < 1) ? 0 : x_vec[i-1];
    const float right = (i < d_vec_size) ? x_vec[i+1] : 0;

    if (i < d_vec_size)
    {
        float result = x_vec[i] * -2;
        result += left + right;
        dest[i] = result;
    }
}

#define WARP_MASK 0xFFFFFFFF

// CUDA kernel that calculates a vector dot product
// NOTE: Destination *must* be zeroed prior to calling
__global__ void parallel_dot(float* dest, const float* a_vec, const float* b_vec)
{   
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane_idx = threadIdx.x & 0x1F;

    if (i >= d_vec_size)
    {
        return;
    }

    float accum = a_vec[i] * b_vec[i];
    
    // Warp-level add-reduce
    accum += __shfl_down_sync(WARP_MASK, accum, 16);
    accum += __shfl_down_sync(WARP_MASK, accum, 8);
    accum += __shfl_down_sync(WARP_MASK, accum, 4);
    accum += __shfl_down_sync(WARP_MASK, accum, 2);
    accum += __shfl_down_sync(WARP_MASK, accum, 1);

    // Accumulate into the global destination buffer, this could be a little more optimal
    // if we reduce into a block shared buffer first, before the global atomic add
    if (lane_idx == 0)
    {
        atomicAdd(dest, accum);
    }
}

// CUDA kernel that calculates the maximum (absolute) value for the whole vector
// NOTE: Destination *must* be zeroed prior to calling
__global__ void parallel_absmax(float* dest, const float* a_vec)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane_idx = threadIdx.x & 0x1F;

    if (i >= d_vec_size)
    {
        return;
    }

    float accum = abs(a_vec[i]);
    
    // Warp-level add-reduce
    accum = max(accum, __shfl_down_sync(WARP_MASK, accum, 16));
    accum = max(accum, __shfl_down_sync(WARP_MASK, accum, 8));
    accum = max(accum, __shfl_down_sync(WARP_MASK, accum, 4));
    accum = max(accum, __shfl_down_sync(WARP_MASK, accum, 2));
    accum = max(accum, __shfl_down_sync(WARP_MASK, accum, 1));

    // Accumulate into the global destination buffer, this could be a little more optimal
    // if we reduce into a block shared buffer first, before the global atomic max
    if (lane_idx == 0)
    {
        atomicAbsMaxFloat(dest, accum);
    }
}

// CUDA kernel that calculates a scaled element-wise vector sum
__global__ void parallel_axby(float* dest, const float* a_vec, const float* x, const float* b_vec, const float* y)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < d_vec_size)
    {
        const float result = a_vec[i] * (*x) + b_vec[i] * (*y);
        dest[i] = result;
    }
}

// CUDA kernel that calculates a single-thread scalar division, with result and negated result.
// Used for calculating the `alpha` and `beta` scalar values on device
__global__ void scalar_div(float* dest, float* neg_dest, const float* x, const float* y)
{
    float result = *x / *y;
    *dest = result;
    *neg_dest = -result;
}

__global__ void fused_matmul_dot(float *matmul_dest, float* dot_dest, const float* p_vec)
{
    const int global_i = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane_i = threadIdx.x & 0x1F;
    const int i = threadIdx.x + 1;

    __shared__ float p_cache[BLOCK_SIZE + 2];
    __shared__ float dot_accum;

    // Init cache for source data
    p_cache[i] = (global_i < d_vec_size) ? p_vec[global_i] : 0;
    // Init boundary values
    if (threadIdx.x == 0)
    {
        dot_accum = 0;
        p_cache[0] = (global_i < 1) ? 0 : p_vec[global_i-1];
        p_cache[BLOCK_SIZE+1] = ((global_i+BLOCK_SIZE) < d_vec_size) ? p_vec[global_i+BLOCK_SIZE] : 0;
    }

    __syncthreads();
    
    float result = 0;
    if (global_i < d_vec_size)
    {
        result += p_cache[i] * -2;
        result += p_cache[i-1];
        result += p_cache[i+1];
        matmul_dest[global_i] = result;
    }

    // Fused wave-reduce dot product with p_vec
    result *= p_cache[i];
    result += __shfl_down_sync(WARP_MASK, result, 16);
    result += __shfl_down_sync(WARP_MASK, result, 8);
    result += __shfl_down_sync(WARP_MASK, result, 4);
    result += __shfl_down_sync(WARP_MASK, result, 2);
    result += __shfl_down_sync(WARP_MASK, result, 1);
    
    if (lane_i == 0)
    {
        atomicAdd(&dot_accum, result);
    }
    
    __syncthreads();

    if (threadIdx.x == 0)
    {
        atomicAdd(dot_dest, dot_accum);
    }
}

__global__ void fused_vecmul_absmax_dot(float *dest_max, float* dest_dot, const float* rdotr, const float* pdotap,
                                        float* x_vec, float* r_vec, const float* p_vec, const float* a_mul_p)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const int lane_i = threadIdx.x & 0x1F;
    const float alpha = *rdotr / *pdotap;

    __shared__ float dot_accum;
    __shared__ float max_accum;

    if (threadIdx.x == 0)
    {
        dot_accum = 0;
        max_accum = 0;
    }

    __syncthreads();

    float r = 0;
    if (i < d_vec_size)
    {   
        const float x = x_vec[i] + p_vec[i] * alpha; 
        x_vec[i] = x;

        r = r_vec[i] - a_mul_p[i] * alpha;
        r_vec[i] = r;
    }

    float r_max = abs(r);
    float r_dot = r * r;
    
    r_max = max(r_max, __shfl_down_sync(WARP_MASK, r_max, 16));
    r_dot += __shfl_down_sync(WARP_MASK, r_dot, 16);

    r_max = max(r_max, __shfl_down_sync(WARP_MASK, r_max, 8));
    r_dot += __shfl_down_sync(WARP_MASK, r_dot, 8);

    r_max = max(r_max, __shfl_down_sync(WARP_MASK, r_max, 4));
    r_dot += __shfl_down_sync(WARP_MASK, r_dot, 4);

    r_max = max(r_max, __shfl_down_sync(WARP_MASK, r_max, 2));
    r_dot += __shfl_down_sync(WARP_MASK, r_dot, 2);

    r_max = max(r_max, __shfl_down_sync(WARP_MASK, r_max, 1));
    r_dot += __shfl_down_sync(WARP_MASK, r_dot, 1);

    if (lane_i == 0)
    {
        atomicAdd(&dot_accum, r_dot);
        atomicAbsMaxFloat(&max_accum, r_max);
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        atomicAdd(dest_dot, dot_accum);
        atomicAbsMaxFloat(dest_max, max_accum);
    }
}

__global__ void fused_beta_finalize(const float* rdotr, const float* new_rdotr, float *p_vec, const float* r_vec)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    const float beta = *new_rdotr / *rdotr;

    if (i < d_vec_size)
    {
        p_vec[i] = r_vec[i] + p_vec[i] * beta;
    }
}

hipError_t verify(hipError_t result)
{
#ifndef NDEBUG
    if (result != hipSuccess)
    {
        std::cout << "CUDA error: " << hipGetErrorString(result) << std::endl;
        assert(!"CUDA call failed");
    }
#endif

    return result;   
}

#define USE_FUSED_KERNELS 1
#define LAUNCH_DOMAIN     div_round_up(size, BLOCK_SIZE), BLOCK_SIZE

bool parallel_conj_grad(const vector<float>& in_b_vec, vector<float>& out_x_vec, const float threshold, const int max_iters)
{
    if (in_b_vec.size() >= INT_MAX)
    {
        return false;
    }
    int size = static_cast<int>(in_b_vec.size());
    verify(hipMemcpyToSymbol(HIP_SYMBOL(d_vec_size), &size, sizeof(d_vec_size)));

    float* d_x_vec;
    float* d_r_vec;
    float* d_p_vec;
    float* d_a_mul_p;
    float* d_result;
    verify(hipMalloc(&d_x_vec, size * sizeof(float)));
    verify(hipMalloc(&d_r_vec, size * sizeof(float)));
    verify(hipMalloc(&d_p_vec, size * sizeof(float)));
    verify(hipMalloc(&d_a_mul_p, size * sizeof(float)));
    verify(hipMalloc(&d_result, size * sizeof(float)));

    float* d_one;
    float* d_neg_one;
    float* d_alpha;
    float* d_neg_alpha;
    float* d_beta;
    float* d_rdotr;
    float* d_new_rdotr;
    float* d_p_dot_a_p;
    float* d_abs_max;
    verify(hipMalloc(&d_one, sizeof(float)));
    verify(hipMalloc(&d_neg_one, sizeof(float)));
    verify(hipMalloc(&d_alpha, sizeof(float)));
    verify(hipMalloc(&d_neg_alpha, sizeof(float)));
    verify(hipMalloc(&d_beta, sizeof(float)));
    verify(hipMalloc(&d_rdotr, sizeof(float)));
    verify(hipMalloc(&d_new_rdotr, sizeof(float)));
    verify(hipMalloc(&d_p_dot_a_p, sizeof(float)));
    verify(hipMalloc(&d_abs_max, sizeof(float)));

    const float one = 1.0;
    const float neg_one = -1.0;

    verify(hipMemset(d_x_vec, 0, size * sizeof(float)));
    verify(hipMemset(d_a_mul_p, 0, size * sizeof(float)));
    verify(hipMemset(d_rdotr, 0, sizeof(float)));

    verify(hipMemcpy(d_alpha, &one, sizeof(float), hipMemcpyHostToDevice));
    verify(hipMemcpy(d_one, &one, sizeof(float), hipMemcpyHostToDevice));
    verify(hipMemcpy(d_neg_one, &neg_one, sizeof(float), hipMemcpyHostToDevice));
    verify(hipMemcpy(d_r_vec, in_b_vec.data(), size * sizeof(float), hipMemcpyHostToDevice));
    verify(hipMemcpy(d_p_vec, d_r_vec, size * sizeof(float), hipMemcpyDeviceToDevice));

    parallel_dot<<<LAUNCH_DOMAIN>>>(d_rdotr, d_r_vec, d_r_vec);

    for (int k = 0; k < max_iters; k++)
    {
        verify(hipMemset(d_p_dot_a_p, 0, sizeof(float)));
        verify(hipMemset(d_abs_max, 0, sizeof(float)));
        verify(hipMemset(d_new_rdotr, 0, sizeof(float)));

#if USE_FUSED_KERNELS
        fused_matmul_dot<<<LAUNCH_DOMAIN>>>(d_a_mul_p, d_p_dot_a_p, d_p_vec);
        fused_vecmul_absmax_dot<<<LAUNCH_DOMAIN>>>(d_abs_max, d_new_rdotr, d_rdotr, d_p_dot_a_p, d_x_vec, d_r_vec, d_p_vec, d_a_mul_p);
        fused_beta_finalize<<<LAUNCH_DOMAIN>>>(d_rdotr, d_new_rdotr, d_p_vec, d_r_vec);
        verify(hipMemcpyAsync(d_rdotr, d_new_rdotr, sizeof(float), hipMemcpyDeviceToDevice));
#else
        parallel_sp_matmul<<<LAUNCH_DOMAIN>>>(d_a_mul_p, d_p_vec); 

        parallel_dot<<<LAUNCH_DOMAIN>>>(d_p_dot_a_p, d_p_vec, d_a_mul_p);
        scalar_div<<<1, 1>>>(d_alpha, d_neg_alpha, d_rdotr, d_p_dot_a_p);

        parallel_axby<<<LAUNCH_DOMAIN>>>(d_x_vec, d_x_vec, d_one, d_p_vec, d_alpha);
        parallel_axby<<<LAUNCH_DOMAIN>>>(d_r_vec, d_r_vec, d_one, d_a_mul_p, d_neg_alpha);

        parallel_absmax<<<LAUNCH_DOMAIN>>>(d_abs_max, d_r_vec);
#endif // USE_FUSED_KERNELS

        float abs_max = 0;
        verify(hipMemcpy(&abs_max, d_abs_max, sizeof(abs_max), hipMemcpyDeviceToHost));
        if (abs_max <= threshold)
        {
            // We've reached an approximate solution within threshold
            out_x_vec.resize(size);
            verify(hipMemcpy(out_x_vec.data(), d_x_vec, size * sizeof(float), hipMemcpyDeviceToHost));
            return true;
        }

#if !USE_FUSED_KERNELS
        parallel_dot<<<LAUNCH_DOMAIN>>>(d_new_rdotr, d_r_vec, d_r_vec);

        // NOTE: `d_neg_alpha` is used as a dummy destination here
        scalar_div<<<1, 1>>>(d_beta, d_neg_alpha, d_new_rdotr, d_rdotr);
        verify(hipMemcpyAsync(d_rdotr, d_new_rdotr, sizeof(float), hipMemcpyDeviceToDevice));

        parallel_axby<<<LAUNCH_DOMAIN>>>(d_p_vec, d_r_vec, d_one, d_p_vec, d_beta);
#endif // USE_FUSED_KERNELS
    }

    return false;
}



